
//GPU implementation of sliding window BP alg.
//for BC decoding L=W=3


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>
#include <time.h>
#include <omp.h>
#include <fstream>
#include <sstream>
#include <random>
#include <chrono>
using namespace std;

float *H,*H2,*H3,*x,*x_hat,*y,P,*LR,*pLR,*E_c_v,*E_v_c,*H_D,*y_D,*LR_D,*pLR_D,*E_c_v_D,*E_v_c_D,err,err2,err3,biterr,biterr2,biterr3,varn; //n is source length, m is code length
long m,n,n2,*vns,*cns,*vns_D,*cns_D,BC;
long col_wt,row_wt,num,dcw,dcw2,dcw3,hshft,vshft,mem,W,call,flg; //row_wt is the max, row weight in case of SC codes
long L=99; //coupling length (multiple of mem+1 and >=W) 

#define CW 2048 //no. of codeword streams per transmission
#define I 50 //no. of iters
#define ev 1 //min. no. of error events

string func(long n) {
	stringstream result;
	result << n;
	return result.str();
}






__global__ void init1(float *LR, float *pLR, float *y, long m, long n, float varn, long num, long hshft, long col_wt, float *E_c_v) {

	long cw = blockIdx.y * blockDim.x + threadIdx.x;
	long i,k,strt,stp;
	
	//printf("cw: %d\n", cw);

	if(cw<CW) {
		if(num==0) //if no shift
			for(i=0;i<n;i++) {
				LR[cw*n+i]=2*y[cw*n+i]/varn; //AWGN case, LLr= ln(Pr[x=0|y]/Pr[x=1|y])= ln [exp{-(y-1)^2/2sig^2)}/exp{-(y-(-1))^2/2sig^2)}], x=0,1 means y=1,-1
				pLR[cw*n+i]=0;
			}
		else {
			//shifting the msgs (from bottom to top). For SC codes, n is window len.
			for(i=0;i<n-hshft;i++) {
				LR[cw*n+i]=LR[cw*n+i+hshft]; //hshft is the no. of new VNs entering window
				pLR[cw*n+i]=pLR[cw*n+i+hshft];
				for(k=0;k<col_wt;k++)
					//E_c_v[i][k][cw]=E_c_v[i+hshft][k][cw]; 
					E_c_v[CW*(col_wt*i+k)+cw]=E_c_v[CW*(col_wt*(i+hshft)+k)+cw];
			}

			strt=num*hshft;
			stp=strt+hshft;

			for(i=strt;i<stp;i++) 
				//LR[(n-hshft)+(i-strt)][cw]=2*y[i]/varn
				LR[cw*n+(n-hshft)+(i-strt)]=2*y[cw*n+i]/varn; //new LLR values entering the window

		} 
	}

	//printf("LR: \n"); for(i=0;i<n;i++) printf(" %f",LR[cw*n+i]);
	//printf("y: \n"); for(i=0;i<n;i++) printf(" %f",y[cw*n+i]);
}





__global__ void init2(long *vns, float *LR, float *E_v_c, long m, long n, long row_wt, long num, long vshft) {

	long cw = blockIdx.y * blockDim.x + threadIdx.x; 
	long j,k;

	//printf("cw= %d\n", cw);

	if(cw<CW) {
		if(num==0) //if no shifts
			//for(j=0;j<m;j++) for(k=0;k<row_wt;k++) E_v_c[j][k][cw]=LR[cw][vns[j][k]];
			for(j=0;j<m;j++) 
				for(k=0;k<row_wt;k++) 
					//E_v_c[j][k][cw]=LR[vns[j][k]][cw];
					E_v_c[CW*(row_wt*j+k)+cw]=LR[cw*n+vns[j*row_wt+k]]; //LLR input

		else {
			//shifting the msgs (from bottom to top). For SC codes, n is window len.
			for(j=0;j<m-vshft;j++) {
				for(k=0;k<row_wt;k++)
					//E_v_c[j][k][cw]=E_v_c[j+vshft][k][cw]; //vshft is the no. of new CNs entering window
					E_v_c[CW*(row_wt*j+k)+cw]=E_v_c[CW*(row_wt*(j+vshft)+k)+cw];
			}

			for(j=0;j<vshft;j++) 
				for(k=0;k<row_wt;k++)
					//E_v_c[j+m-vshft][k][cw]=LR[vns[j+m-vshft][k]][cw]; 
					E_v_c[CW*(row_wt*(j+m-vshft)+k)+cw]=LR[cw*n+vns[(j+m-vshft)*row_wt+k]]; //new CNs of window initialized with new LLR values 

		}
	}

	//printf("E_v_c: \n"); for(j=0;j<n;j++) printf(" %.1f",E_v_c[j]);
	//printf("num= %d\n", num);
}








__global__ void horz(long *vns, long *cns, float *E_v_c, float *E_c_v, long m, long n, long row_wt, long col_wt) {
	long i,k,vidx;
	float tmp;

	long cw = blockIdx.y * blockDim.x + threadIdx.x;
	long j = blockIdx.x;

	//printf("cw,j=%d,%d\n", cw,j);
	//printf("\n");

	if(cw<CW) {
		if(j<m) { //m is no. of CNs
			for(i=0;i<row_wt;i++) { //row_wt is the no. of neighboring VNs of CN j
				vidx=vns[j*row_wt+i]; //index of ith neighboring VN of CN j

				if(vidx>-1) {
					tmp=1; 
					for(k=0;k<row_wt;k++) 
						if(vns[j*row_wt+k]>-1 && k!=i) tmp*=tanh(0.5*E_v_c[CW*(row_wt*j+k)+cw]); //jth CN accumulating msgs from all neighboring VNs except i
					
					//cout<<'\n'<<"j: "<<j<<" vidx: "<<vidx<<" tmp: "<<tmp<<endl;
		
					for(k=0;k<col_wt;k++) 
						if(cns[vidx*col_wt+k]==j) {E_c_v[CW*(vidx*col_wt+k)+cw]=2*atanhf(tmp); break;} //msg sent by jth CN to ith VN				
				}			
			} 
		}
	}
	
}







__global__ void vert(long *cns, long *vns, float *LR, float *pLR, float *E_v_c, float *E_c_v, long m, long n, long col_wt, long row_wt) {
	long j,k,cidx;
	float tmp;

	long cw = blockIdx.y * blockDim.x + threadIdx.x;
	long i = blockIdx.x;

	//printf("cw,i=%d,%d\n", cw,i);

	if(cw<CW) {
		if(i<n) { //n is no. of VNs
			for(j=0;j<col_wt;j++) { //col_wt is the no. of neighboring CNs of VN i
				cidx=cns[i*col_wt+j];  //index of jth neighboring CN of VN i

				if(cidx>-1) {
					tmp=0; 
					for(k=0;k<col_wt;k++) 
						if(cns[i*col_wt+k]>-1 && k!=j) tmp+=E_c_v[CW*(col_wt*i+k)+cw]; //ith VN accumulating msg from all neighboring CNs except j 
	
					for(k=0;k<row_wt;k++) 
						if(vns[cidx*row_wt+k]==i) {E_v_c[CW*(row_wt*cidx+k)+cw]=tmp+LR[cw*n+i]; break;} //msg sent by ith VN to jth CN
						//printf("tmp2=%f\n",tmp2);					
				}
			} 

			//updating the aposteriori LLR

			tmp=0; 
			for(k=0;k<col_wt;k++) 
				if(cns[col_wt*i+k]>-1) tmp+=E_c_v[CW*(col_wt*i+k)+cw];
			pLR[cw*n+i]=LR[cw*n+i]+tmp; 
 
		}
	}
}





void BP() {
	long i,cw,j,l,strt=0,stp,STP,flg3;

	//E_v_c[j][i][cw]: msg accumulated by CN j from VN i for codeword stream cw, E_v_c[j][i][cw]=E_v_c[CW*(no. of cols *j + i)+cw]
	//E_c_v[i][j][cw]: msg accumulated by VN i from CN j for codeword stream cw

	//testing
	//dim3 dimBlock(n,CW); //n (CW) threads in a block in the x (y) dimension (max. 1024 threads in product of x and y dimension). Think of coordinates on Cartesian plane
	//dim3 dimGrid(1,1); //no. of blocks in x and y dimension

	long a=1024,b=1; //1024; //no. of threads per block=a*b, and max 1024

	dim3 dimBlock(a,b); 
	dim3 dimGrid(1,CW/a); 
	
	dim3 dimBlock4(a,b); 
	dim3 dimGrid4(1,CW/a); //no. of blocks in x and y dimension
	
	//*********************** horz
	dim3 dimBlock2(a,b); 
	dim3 dimGrid2(m,CW/a); 

	//*********************** vert
	dim3 dimBlock3(a,b); 
	dim3 dimGrid3(n,CW/a); 

	//err calculation
	/*dim3 dimBlock5(a,b); 
	dim3 dimGrid5(1,CW/a);*/



	num=0;
	while(num<=(L-W)/(mem+1)) {

		//cout<<'\n'<<"num: "<<num<<endl;
		//initializing the window
		init1<<<dimGrid,dimBlock>>>(LR_D,pLR_D,y_D,m,n,varn,num,hshft,col_wt,E_c_v_D); 
		hipDeviceSynchronize(); 

		init2<<<dimGrid4,dimBlock4>>>(vns_D,LR_D,E_v_c_D,m,n,row_wt,num,vshft); 
		hipDeviceSynchronize(); 

		for(l=0;l<I;l++) {
			//horizontal step
			horz<<<dimGrid2,dimBlock2>>>(vns_D,cns_D,E_v_c_D,E_c_v_D,m,n,row_wt,col_wt);
			hipDeviceSynchronize(); 
		
			//vertical step
			vert<<<dimGrid3,dimBlock3>>>(cns_D,vns_D,LR_D,pLR_D,E_v_c_D,E_c_v_D,m,n,col_wt,row_wt); 
			hipDeviceSynchronize(); 		
		
		} 
		hipMemcpy(pLR,pLR_D,CW*n*sizeof(float),hipMemcpyDeviceToHost);

		//cout<<'\n'<<"pLR: "; for(cw=0;cw<CW;cw++){ for(i=0;i<n;i++) cout<<pLR[cw*n+i]<<" "; cout<<'\n'<<'\n';}
		
		//hard decision
		if(num<(L-W)/(mem+1) || BC) STP=1;
		else STP=W/(mem+1); //decision in the last window position

		for(j=0;j<STP;j++) {
			if(num<(L-W)/(mem+1))
				strt=num*hshft;
			else if(num==0 && j==0) 
				strt=0;
			else strt+=hshft;

			stp=strt+hshft;
			//cout<<'\n'<<"strt: "<<strt<<" stp: "<<stp<<endl;
			
			for(cw=0;cw<CW;cw++) {
				flg3=0;
				for(i=strt;i<stp;i++) {
					if(pLR[cw*n+i-strt]<0) {
						if(!flg3) {
							if(!flg) err++; //for blk err
							else if(flg==1) err2++;
							else if(flg==2) err3++;
							flg3=1;
						}
						if(!flg) biterr++; //for bit err
						else if(flg==1) biterr2++;
						else if(flg==2) biterr3++;
					}
 					//if(pLR[cw*n+i-strt]>=0) 
						//x_hat[cw*n+i]=0; 
					//else 
						//x_hat[cw*n+i]=1;
				}
				if(!flg) dcw++; //no. of decoded cws
				else if(flg==1) dcw2++;
				else if(flg==2) dcw3++;

				//for(i=strt;i<stp;i++) 
					//if(x[cw*n+i]!=x_hat[cw*n+i]) {
						//err++; 
						//break;
				//}	
			}
		
			//cout<<'\n'<<"pLR: "; for(cw=0;cw<CW;cw++){ for(i=strt;i<stp;i++) cout<<pLR[cw*n+i-strt]<<" "; cout<<'\n'<<'\n';}

		}
		num++; //no. of window shifts

		if(!flg && err>=ev) break; //to avoid waiting until the last window position
		else if(flg==1 && err2>=ev) break;
		else if(flg==2 && err3>=ev) break;
		
	}
		
}	


int main() {	
	srand(time(0));	
	clock_t tStart = clock();	
	long i,i2,j,kk; 
	long fn,num_dat,cnt,cw,gama,p,J; 

	float *blk_err,*blk_err2,*blk_err3,*bit_err,*bit_err2,*bit_err3,R;  
	cout<<'\n'<<" device no: "; cin>>fn; ///////////////////////////////// 
	cout<<'\n'<<" kk: "; cin>>kk; 
	if(!kk) { //if not Kelley Kliewer code
		cout<<'\n'<<" col_wt: "; cin>>col_wt; 
		cout<<'\n'<<" BC?: "; cin>>BC; //1 for BC, 0 for SC
	}
	else BC=1;

	//fn=1;
	hipSetDevice(fn-1); //select GPU card (0 or 1)	

	//construct a random generator engine:
	std::random_device rd;
    	std::mt19937 e2(rd());

	ifstream inf,inf1,inf2,inf3,inf4,inf5,inf6,inf7,inf8,inf9,inf10,inf11; 
	if(!BC && !kk) {
		inf1.open("mval.txt"); inf1>>m; 
		inf2.open("nval.txt"); inf2>>n; //window length
	} 
	else if(BC && !kk) {
		inf1.open("mval_unc.txt"); inf1>>m; 
		inf2.open("nval_unc.txt"); inf2>>n;
	} 
	else if(kk) {
		inf1.open("mval_kk.txt"); inf1>>m; 
		inf2.open("nval_kk.txt"); inf2>>n;
	}

	inf3.open("J.txt"); inf3>>J; 
	inf4.open("p.txt"); inf4>>p; 
	inf5.open("mem.txt"); inf5>>mem;
	inf6.open("gama.txt"); inf6>>gama;
	inf7.open("W.txt"); inf7>>W; //no. of col. blks of window (multiple of mem+1). width of col. blk is Jp^2
	if(BC) {L=3;W=3;}

	//code rate
	if(BC) R=1-float(m)/float(n);
	else R=1-float(col_wt)/float(p)*(1+float(mem)/float(L));

	H= new float[m*n]; //H is the window. Since windows are identical, only the window Tanner graph is needed. The Lvalues "move" across the window
	H2= new float[m*n];
	H3= new float[m*n];

	//inf.open("H_3_7_24_5_sc_rnd1.txt"); for(j=0;j<m;j++) for(i=0;i<n;i++) inf >> H[j*n+i]; //upload the parity-check matrix
	//inf.open("mat_p5_sc.txt"); for(j=0;j<m;j++) for(i=0;i<n;i++) inf >> H[j*n+i];

	if(kk) { //Kelley Kliewer random lifted BC
		inf.open("mat_kk.txt"); for(j=0;j<m;j++) for(i=0;i<n;i++) inf >> H[j*n+i];
		inf9.open("col_wt_kk.txt"); inf9 >> col_wt; //maximum values
		inf10.open("row_wt_kk.txt"); inf10 >> row_wt;
	}
	else if(col_wt==3 && !BC) {
		inf.open("win_3_sc_rnd1.txt"); for(j=0;j<m;j++) for(i=0;i<n;i++) inf >> H[j*n+i];
		inf8.open("win_3_sc_M1.txt"); for(j=0;j<m;j++) for(i=0;i<n;i++) inf8 >> H2[j*n+i]; //method 1
		inf11.open("win_3_sc_M2.txt"); for(j=0;j<m;j++) for(i=0;i<n;i++) inf11 >> H3[j*n+i]; //method 2
	}
	else if(col_wt==3 && BC) {
		inf.open("win_3_unc.txt"); for(j=0;j<m;j++) for(i=0;i<n;i++) inf >> H[j*n+i];
	}
	else if(col_wt==4 && !BC) {
		inf.open("win_4_sc_rnd1.txt"); for(j=0;j<m;j++) for(i=0;i<n;i++) inf >> H[j*n+i];
		inf8.open("win_4_sc_M1.txt"); for(j=0;j<m;j++) for(i=0;i<n;i++) inf8 >> H2[j*n+i];
		inf11.open("win_4_sc_M2.txt"); for(j=0;j<m;j++) for(i=0;i<n;i++) inf11 >> H3[j*n+i];
	}
	else if(col_wt==4 && BC) {
		inf.open("win_4_unc.txt"); for(j=0;j<m;j++) for(i=0;i<n;i++) inf >> H[j*n+i];
	}
	if(!kk) row_wt=p;

	//cout<<'\n'<<"H: "<<'\n'; for(j=0;j<m;j++){for(i=0;i<n;i++) {cout<<H[j*n+i]<<" "; if(i>0 && (i+1)%(row_wt)==0) cout<<"  ";} cout<<'\n'; if(j>0 && (j+1)%(row_wt)==0) cout<<'\n';}

	//col_wt=0;
	//for(j=0;j<m;j++) if(H[j*n+0]) col_wt++;

	cout<<'\n'<<"m: "<<m<<" n: "<<n<<" R: "<<R<<" col_wt: "<<col_wt<<" row_wt: "<<row_wt<<" gama: "<<gama<<" W: "<<W<<endl; 

	//here n=WJp^2
	
	//these don't affect BC or kk code
	if(!BC) {
		n2=L*J*p*p; //length of codestream
		hshft=J*p*p*(mem+1); //amount of right shift of the window (length of a CW in the stream)
		vshft=gama*p*J*(mem+1); //amount of downward shift
	}
	else {
		n2=n;
		hshft=n;
		vshft=m;
	}

	cout<<'\n'<<"hshft: "<<hshft<<" vshft: "<<vshft<<" n2: "<<n2<<endl; 
	
	//2D arrays
	//x= new float[CW*n2]; //each row has a different CW stream
	//x_hat= new float[CW*n2];
	y= new float[CW*n2];  
	//LR= new float[CW*n];  //Lvalues in the window
	pLR= new float[CW*n];  

	vns=new long[m*row_wt]; //all VNs of CN j in a row
	
	//3D arrays
	//E_v_c= new float[m*row_wt*CW];
	//E_c_v= new float[n*col_wt*CW];

	cns=new long[n*col_wt]; //all CNs of a VN in a row

	//allocating corresponding matrices on device
	//cudaMalloc((void**)&H_D,m*n*sizeof(float));
	hipMalloc((void**)&y_D,CW*n2*sizeof(float));
	hipMalloc((void**)&vns_D,m*row_wt*sizeof(long)); 
	hipMalloc((void**)&cns_D,n*col_wt*sizeof(long));
	hipMalloc((void**)&LR_D,CW*n*sizeof(float));
	hipMalloc((void**)&pLR_D,CW*n*sizeof(float));
	hipMalloc((void**)&E_v_c_D,m*row_wt*CW*sizeof(float));
	hipMalloc((void**)&E_c_v_D,n*col_wt*CW*sizeof(float));



	//*********************col_wt 3****************************
	//Eb/No values in dB
	float EbNo[]={0,0.5,1,1.25,1.5,1.75,2,2.2,2.3,2.4,2.5}; //p=7, SC 
 
	float EbNo_BC[]={0,0.5,1,1.25,1.5,1.75,2,2.2,2.3,2.4,2.5}; //p=7, BC 
	
	//*********************col_wt 4****************************
	//float EbNo2[]={0,0.25,0.5,0.75,1,1.2,1.4,1.6,1.7,1.8,1.9,2,2.1,2.2,2.3}; //p=7, SC
	float EbNo2[]={2.4}; //p=7, SC

	//float EbNo2_BC[]={0,0.25,0.5,0.75,1,1.2,1.4,1.6,1.7,1.8,1.9,2,2.1,2.2}; //p=7, BC
	float EbNo2_BC[]={0,0.25,0.5,0.75,1,1.2,1.4,1.6,2.3,2.4};


	float EbNo_kk[]={0,0.4,0.8,1.15,1.45,1.8,2,2.5}; //for Kelley Kliewer random sub-code1
	//float EbNo_kk[]={0,0.5,1,1.2,1.3,1.4,1.5,1.6,1.7,1.8,1.9}; //(3,6) LDPC BC, n=4098 

	if(!kk) {
		if(col_wt==3 && !BC) num_dat=sizeof(EbNo)/sizeof(EbNo[0]);
		else if(col_wt==3 && BC) num_dat=sizeof(EbNo_BC)/sizeof(EbNo_BC[0]);
		else if(col_wt==4 && !BC) num_dat=sizeof(EbNo2)/sizeof(EbNo2[0]);
		else if(col_wt==4 && BC) num_dat=sizeof(EbNo2_BC)/sizeof(EbNo2_BC[0]);
	}
	else num_dat=sizeof(EbNo_kk)/sizeof(EbNo_kk[0]);

	blk_err= new float[num_dat]; blk_err2= new float[num_dat]; blk_err3= new float[num_dat];
	bit_err= new float[num_dat]; bit_err2= new float[num_dat]; bit_err3= new float[num_dat]; 


	if(BC) {
		//initializing
		for(i=0;i<m;i++) for(j=0;j<row_wt;j++) vns[i*row_wt+j]=-1;
		for(i=0;i<n;i++) for(j=0;j<col_wt;j++) cns[i*col_wt+j]=-1;

		for(j=0;j<m;j++) {
			cnt=0; 
			for(i=0;i<n;i++) if(H[j*n+i]) {vns[j*row_wt+cnt]=i; cnt++;} 
		}
		//cout<<'\n'<<"vns "<<'\n'; for(i=0;i<m;i++) {for(j=0;j<row_wt;j++) cout<<vns[i*row_wt+j]<<" "; cout<<'\n';}
		for(i=0;i<n;i++) {
			cnt=0; 
			for(j=0;j<m;j++) if(H[j*n+i]) {cns[i*col_wt+cnt]=j; cnt++;} 
		}
		//cout<<'\n'<<"cns "<<'\n'; for(i=0;i<n;i++) {for(j=0;j<col_wt;j++) cout<<cns[i*col_wt+j]<<" "; cout<<'\n';}
		hipMemcpy(vns_D,vns,m*row_wt*sizeof(long),hipMemcpyHostToDevice);
		hipMemcpy(cns_D,cns,n*col_wt*sizeof(long),hipMemcpyHostToDevice);
	}
	

	for(i2=0;i2<num_dat;i2++) {
		if(!kk) {
			if(col_wt==3 && !BC) varn=1/(2*R*pow(10,0.1*EbNo[i2]));
			else if(col_wt==3 && BC) varn=1/(2*R*pow(10,0.1*EbNo_BC[i2]));
			else if(col_wt==4 && !BC) varn=1/(2*R*pow(10,0.1*EbNo2[i2]));
			else if(col_wt==4 && BC) varn=1/(2*R*pow(10,0.1*EbNo2_BC[i2]));
		}
		else varn=1/(2*R*pow(10,0.1*EbNo_kk[i2])); //if EbNo not in dB, then varn=1/(2*R*EbNo_kk[i2]);

		std::normal_distribution<float> dist(0,sqrt(varn)); //should be s.d. not variance

		err=err2=err3=biterr=biterr2=biterr3=dcw=dcw2=dcw3=call=0;
		while((!BC && (err<ev || err2<ev || err3<ev)) || (BC && err<ev)) { //take avg. of min. 3 error events

			for(cw=0;cw<CW;cw++) for(i=0;i<n2;i++) y[cw*n2+i]=1+dist(e2); //adding gaussian noise to all-zero CW
			hipMemcpy(y_D,y,CW*n2*sizeof(float),hipMemcpyHostToDevice);

			flg=0;
			while((!BC && flg<3) || (BC && !flg)) {

				if(!flg && !BC) {
					//initializing
					for(i=0;i<m;i++) for(j=0;j<row_wt;j++) vns[i*row_wt+j]=-1;
					for(i=0;i<n;i++) for(j=0;j<col_wt;j++) cns[i*col_wt+j]=-1;

					for(j=0;j<m;j++) {
						cnt=0; 
						for(i=0;i<n;i++) if(H[j*n+i]) {vns[j*row_wt+cnt]=i; cnt++;} 
					}
					//cout<<'\n'<<"vns "<<'\n'; for(i=0;i<m;i++) {for(j=0;j<row_wt;j++) cout<<vns[i*row_wt+j]<<" "; cout<<'\n';}
	
					for(i=0;i<n;i++) {
						cnt=0; 
						for(j=0;j<m;j++) if(H[j*n+i]) {cns[i*col_wt+cnt]=j; cnt++;} 
					}
					//cout<<'\n'<<"cns "<<'\n'; for(i=0;i<n;i++) {for(j=0;j<col_wt;j++) cout<<cns[i*col_wt+j]<<" "; cout<<'\n';}
				}
				else if(flg==1 && !BC) {
					//initializing
					for(i=0;i<m;i++) for(j=0;j<row_wt;j++) vns[i*row_wt+j]=-1;
					for(i=0;i<n;i++) for(j=0;j<col_wt;j++) cns[i*col_wt+j]=-1;

					for(j=0;j<m;j++) {
						cnt=0; 
						for(i=0;i<n;i++) if(H2[j*n+i]) {vns[j*row_wt+cnt]=i; cnt++;} 
					}
					//cout<<'\n'<<"vns "<<'\n'; for(i=0;i<m;i++) {for(j=0;j<row_wt;j++) cout<<vns[i*row_wt+j]<<" "; cout<<'\n';}
	
					for(i=0;i<n;i++) {
						cnt=0; 
						for(j=0;j<m;j++) if(H2[j*n+i]) {cns[i*col_wt+cnt]=j; cnt++;} 
					}
					//cout<<'\n'<<"cns "<<'\n'; for(i=0;i<n;i++) {for(j=0;j<col_wt;j++) cout<<cns[i*col_wt+j]<<" "; cout<<'\n';}
				}
				else if(flg==2 && !BC) {
					//initializing
					for(i=0;i<m;i++) for(j=0;j<row_wt;j++) vns[i*row_wt+j]=-1;
					for(i=0;i<n;i++) for(j=0;j<col_wt;j++) cns[i*col_wt+j]=-1;

					for(j=0;j<m;j++) {
						cnt=0; 
						for(i=0;i<n;i++) if(H3[j*n+i]) {vns[j*row_wt+cnt]=i; cnt++;} 
					}
					//cout<<'\n'<<"vns "<<'\n'; for(i=0;i<m;i++) {for(j=0;j<row_wt;j++) cout<<vns[i*row_wt+j]<<" "; cout<<'\n';}
	
					for(i=0;i<n;i++) {
						cnt=0; 
						for(j=0;j<m;j++) if(H3[j*n+i]) {cns[i*col_wt+cnt]=j; cnt++;} 
					}
					//cout<<'\n'<<"cns "<<'\n'; for(i=0;i<n;i++) {for(j=0;j<col_wt;j++) cout<<cns[i*col_wt+j]<<" "; cout<<'\n';}
				}
				
				if(!BC) {
					hipMemcpy(vns_D,vns,m*row_wt*sizeof(long),hipMemcpyHostToDevice);
					hipMemcpy(cns_D,cns,n*col_wt*sizeof(long),hipMemcpyHostToDevice);
				}

				BP(); //call wind. dec.

				//if(!flg) {cout<<"err: "<<err<<" dcw: "<<dcw<<endl; }
				//else {cout<<"err2: "<<err2<<" dcw2: "<<dcw2<<endl; cout<<'\n';}

				//if(!flg) call++;	
			
				flg++;
				//cout<<'\n'<<"call: "<<call<<" flg: "<<flg<<" i2: "<<i2;
			}

		}

		//trans=(call*n2/hshft)*CW; //there are n2/hshft cws in a stream. Each cw has length hshft bits in a stream of length n2 bits
		blk_err[i2]=err/dcw;
		blk_err2[i2]=err2/dcw2;
		blk_err3[i2]=err3/dcw3;
	
		bit_err[i2]=biterr/(hshft*dcw);
		bit_err2[i2]=biterr2/(hshft*dcw2);
		bit_err3[i2]=biterr3/(hshft*dcw3);


		cout<<"EbNo: "; 
		for(i=0;i<=i2;i++) 
			if(!kk) {
				if(col_wt==3 && !BC) cout<<EbNo[i]<<" ";
				else if(col_wt==3 && BC) cout<<EbNo_BC[i]<<" "; 
				else if(col_wt==4 && !BC) cout<<EbNo2[i]<<" "; 
				else if(col_wt==4 && BC) cout<<EbNo2_BC[i]<<" "; 
			}
			else cout<<EbNo_kk[i]<<" ";
		cout<<'\n'<<endl;
	
		cout<<"BLER: "; for(i=0;i<=i2;i++) cout<<blk_err[i]<<" "; cout<<'\n'<<endl;
		cout<<"BER: "; for(i=0;i<=i2;i++) cout<<bit_err[i]<<" "; cout<<'\n'<<endl;

		if(!BC) {
			cout<<"BLER_M1: "; for(i=0;i<=i2;i++) cout<<blk_err2[i]<<" "; cout<<'\n'<<endl;
			cout<<"BER_M1: "; for(i=0;i<=i2;i++) cout<<bit_err2[i]<<" "; cout<<'\n'<<endl;

			cout<<"BLER_M2: "; for(i=0;i<=i2;i++) cout<<blk_err3[i]<<" "; cout<<'\n'<<endl;
			cout<<"BER_M2: "; for(i=0;i<=i2;i++) cout<<bit_err3[i]<<" "; cout<<'\n'<<endl;
		}
		
		 
	}
	
	string filename;
	//cout<<'\n'<<"blk_err: "; for(i=0;i<num_dat;i++) cout<<blk_err[i]<<" ";
	ofstream outf1, outf2, outf3, outf4, outf5, outf6; 
	filename="BER"+func(fn)+".txt"; outf1.open(filename.c_str()/*,fstream::app*/); for(i=0;i<num_dat;i++) outf1<<bit_err[i]<<" ";  outf1<<std::endl; outf1.close();
	filename="BER1"+func(fn)+".txt"; outf2.open(filename.c_str()/*,fstream::app*/); for(i=0;i<num_dat;i++) outf2<<bit_err2[i]<<" ";  outf2<<std::endl; outf2.close();
	filename="BER2"+func(fn)+".txt"; outf3.open(filename.c_str()/*,fstream::app*/); for(i=0;i<num_dat;i++) outf3<<bit_err3[i]<<" ";  outf3<<std::endl; outf3.close();

	filename="BLER"+func(fn)+".txt"; outf4.open(filename.c_str()/*,fstream::app*/); for(i=0;i<num_dat;i++) outf4<<blk_err[i]<<" ";  outf4<<std::endl; outf4.close();
	filename="BLER1"+func(fn)+".txt"; outf5.open(filename.c_str()/*,fstream::app*/); for(i=0;i<num_dat;i++) outf5<<blk_err2[i]<<" ";  outf5<<std::endl; outf5.close();
	filename="BLER2"+func(fn)+".txt"; outf6.open(filename.c_str()/*,fstream::app*/); for(i=0;i<num_dat;i++) outf6<<blk_err3[i]<<" ";  outf6<<std::endl; outf6.close();

	//Free device memory
    	hipFree(y_D);
    	hipFree(vns_D);
	hipFree(cns_D);
	hipFree(LR_D);
	hipFree(pLR_D);
	hipFree(E_v_c_D);
	hipFree(E_c_v_D);

	cout<<'\n';
    	printf("executed in: %.2fs\n", (double)(clock() - tStart)/CLOCKS_PER_SEC);
	cout<<'\n';


}





